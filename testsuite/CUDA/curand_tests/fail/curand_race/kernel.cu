#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1
//kernel.cu:8:21: write by thread

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] = hiprand_uniform(state);
}
