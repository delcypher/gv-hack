//xfail:BOOGIE_ERROR
//--blockDim=256 --gridDim=2 --no-inline
//Write by thread [\d]+ in block [\d]+, .+kernel\.cu:9:21:

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
  if (threadIdx.x == 0) {
    A[blockIdx.x] = hiprand(state);
  }
}
