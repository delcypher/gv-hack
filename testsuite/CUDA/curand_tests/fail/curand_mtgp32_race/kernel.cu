//xfail:BOOGIE_ERROR
//--blockDim=512 --gridDim=1
//kernel.cu:8:21: write by thread

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
   A[threadIdx.x] = hiprand(state);
}
