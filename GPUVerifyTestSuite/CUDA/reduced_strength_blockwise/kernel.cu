#include "hip/hip_runtime.h"
//pass
//--blockDim=256 --gridDim=2 -DWIDTH=2064
#include <hip/hip_runtime.h>

/*
 * This kernel demonstrates a blockwise strength-reduction loop.
 * Each block is given a disjoint partition (of length WIDTH) of A.
 * Then each thread writes multiple elements in the partition.
 * It is not necessarily the case that WIDTH%blockDim.x == 0
 */

__global__ void k(int *A) {
  __assert(blockDim.x <= WIDTH);

  for (int i=threadIdx.x; 
    // working set using global invariants
    /*A*/__global_invariant(__write_implies(A, (blockIdx.x*WIDTH) <= __write_offset(A)/sizeof(int))),
    /*B*/__global_invariant(__write_implies(A,                       __write_offset(A)/sizeof(int) < (blockIdx.x+1)*WIDTH)),
    /*C*/__invariant(threadIdx.x <= i),
    /*D*/__invariant(               i <= WIDTH+blockDim.x),

  //// working set iff WIDTH % blockDim.x == 0
  ///*A*/__invariant(__write_implies(A, (blockIdx.x*WIDTH) <= __write_offset(A)/sizeof(int))),
  ///*B*/__invariant(__write_implies(A,                       __write_offset(A)/sizeof(int) < (blockIdx.x+1)*WIDTH)),
  ///*C*/__invariant(threadIdx.x <= i),
  ///*D*/__invariant(               i <= WIDTH+blockDim.x),
  ///*E*/__invariant(__uniform_int((i-threadIdx.x))),
  ///*F*/__invariant(__uniform_bool(__enabled())),

    i<WIDTH; i+=blockDim.x) {
    A[blockIdx.x*WIDTH+i] = i;
  }

#ifdef FORCE_FAIL
  __assert(false);
#endif
}
